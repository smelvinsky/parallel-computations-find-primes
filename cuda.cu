/**
 *  @file   cuda.cu
 *  @author Damian Smela <damian.a.smela@gmail.com>
 *  @date   10.02.2019
 *  @brief  CUDA example - finds all the primes in the large list of positive 
 *          integers.
 */

/********************************* INCLUDES ***********************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <memory.h>

/********************************* DEFINES ************************************/

#define FILE_LINE_BUFF_SIZE (128U)
#define TMP_STR_BUFF_SIZE   (FILE_LINE_BUFF_SIZE)
#define LIST_FILENAME       "prime_list.txt"
#define NUM_OF_THREADS      (4U)

#define NUM_OF_CUDA_BLOCKS  (64U)
#define NUM_OF_CUDA_THREADS (128U)

/***************************** STATIC FUNCTIONS *******************************/

/* Converts ASCII char to corresponding number */
static int ascii_to_num(char ascii_char)
{
    return ((int) ascii_char) - 48;
}


/* Checks if given string is a integer number */
static bool is_number(char num_str[])
{
    char *str_ptr = &num_str[0];

    while(*str_ptr != '\0')
    {
     	if (ascii_to_num(*str_ptr) < 0 || ascii_to_num(*str_ptr) > 9)
        {
            return false;
        }
	    str_ptr++;
    }
    return true;
}


/* Returns the next positive integer from list file */
static int get_next_num_from_file(FILE      *file_ptr, 
                                  char      *line_buff, 
                                  size_t    *line_buff_size)
{
    int num_of_line_chars_tmp;

    memset(line_buff, 0, *line_buff_size);
    num_of_line_chars_tmp = (int) getline(&line_buff, line_buff_size, file_ptr);
    line_buff[num_of_line_chars_tmp - 1] = '\0';                                /* Erase '\n' char from string*/

    if (!is_number(line_buff))
    {
     	return -1;
    }
    else
    {
     	return atoi(line_buff);
    }
}


/* Resolve number of CUDA blocks and threads to cover all the remaining ints
 * to process */
static void resolve_cuda_threads(const int  ints_remaining,
                                 int        *num_of_cuda_blocks,
                                 int        *num_of_cuda_threads)
{
    /* Change the defaults only if there is lower number of data to process
     * than threads in a block */
    if (ints_remaining < NUM_OF_CUDA_BLOCKS * NUM_OF_CUDA_THREADS)
    {
        /* Load the defaults */
        *num_of_cuda_blocks = NUM_OF_CUDA_BLOCKS;
        *num_of_cuda_threads = NUM_OF_CUDA_THREADS;

        /* Check if the next iteration can be run in one-shot */
        for (int i = *num_of_cuda_threads; i > 0; i--)
        {
            if (ints_remaining % i == 0)
            {
                /* If there is enough blocks */
                if (ints_remaining / i < NUM_OF_CUDA_BLOCKS)
                {
                    /* The last iteration can be run in a one shot */
                    *num_of_cuda_blocks = ints_remaining / i;
                    *num_of_cuda_threads = i;
                    return;
                }
            }
        }

        /* The code reaches here if there is no way to run the code in
         * a one-shot */
        
        /* Calculate how many full blocks can run the next iteration 
         * and modify the defaults for the next iteration */
        *num_of_cuda_blocks = ints_remaining / NUM_OF_CUDA_THREADS;
    }
}                                                    

/******************************* CUDA KERNELS *********************************/

__global__ void is_prime_kernel(int *int_list)
{
    /* Get the unique thread index */
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    /* Check if the integer of this index is prime */
    for (int i = 2; i < *(int_list + index); i++)
    {
        if (*(int_list + index) % i == 0)
        {
            /* If it's not a prime then change its value to -1 
             * so it wont be written to the file */
            *(int_list + index) = -1;
            break;
        }
    }
}

/*********************************** MAIN *************************************/

int main(int argc, char **argv)
{
    FILE *file_load_ptr = NULL;                                                 /* File pointer for loaded file */
    FILE *file_gen_ptr = NULL;                                                  /* File pointer for generated file */
    char *filename;                                                             /* Pointer to the string containing file name */
    char *line_buff;                                                            /* Buffer storing temp line of a file */
    size_t buff_size = FILE_LINE_BUFF_SIZE;                                     /* Size of a line buffer (can be realloc()-ed by the getline() func) */
    int num_of_line_chars_tmp;                                                  /* Var storing temp num of chars in the line */
    char str_buff_tmp[TMP_STR_BUFF_SIZE];                                       /* String buffer for temp actions */
    int num_of_ints;                                                            /* Number of integers to analyze */

    int num_of_cuda_blocks = NUM_OF_CUDA_BLOCKS;                                /* Number of cuda blocks */
    int num_of_cuda_threads = NUM_OF_CUDA_THREADS;                              /* Number of cuda threads */                     

    int prime_cnt = 0;                                                          /* Number of primes found */

    float time;                                                                 /* Variables used for measuring time */
    hipEvent_t start, stop;                                                    /* - || - */

    printf("-------------------------CUDA Example-------------------------\n");
    printf("Finds all the primes in the large list of positive integers\n\n");

    /* Check if there is exactly one argument given */
    if (argc != 2)
    {
     	printf("Program needs exactly one argument - filename of the list " 
               "containing positive integer numbers!\n");
        return 1;
    }

    filename = argv[1];

    /* Open the ./"filename" file with read permission */
    file_load_ptr = fopen(filename, "r");
    if (file_load_ptr == NULL)
    {
     	printf("Couldn't open \"%s\" file\n", filename);
        return 2;
    }

    /* Allocate buffers for string parsing operations */
    line_buff = (char *) malloc(buff_size);
    memset(line_buff, 0, buff_size);
    memset(str_buff_tmp, 0, TMP_STR_BUFF_SIZE);

    /* Read the first line from file */
    num_of_line_chars_tmp = (int) getline(&line_buff, 
                                          &buff_size, 
                                          file_load_ptr);

    /* Check if the first line fits the "list_len=X" pattern */
    if ((num_of_line_chars_tmp < 11) || 
        (strncmp(line_buff, "list_len=", 9) != 0))
    {
        printf("Error in the first line - \"%s\" not matching" 
               "\"list_len=X\" pattern!\n", line_buff);

        return 3;
    }
    
    /* Check if the first line contains a number of integers */
    strncpy(str_buff_tmp, &line_buff[9], (size_t) num_of_line_chars_tmp - 10);
    if (!is_number(str_buff_tmp))
    {
     	printf("Error in the first line - \"%s\" is not a correct "
               "list length value\n", str_buff_tmp);

        return 4;
    }

    /* Store the number of integers in this variable */
    num_of_ints = atoi(str_buff_tmp);
    printf("Loading list of %d integer numbers...\n", num_of_ints);

    /* Allocate memory for all the integers */
    int *int_list = (int *) malloc(sizeof(int) * num_of_ints);

    /* Fill the memory with the integers from file */
    for (int i = 0; i < num_of_ints; i++)
    {
     	int_list[i] = get_next_num_from_file(file_load_ptr, 
                                             line_buff, 
                                             &buff_size);
    }

    /* Create a "LIST_FILENAME" file with write permission */
    file_gen_ptr  = fopen(LIST_FILENAME, "w");
    if (file_gen_ptr == NULL)
    {
     	printf("Couldn't open/create \"%s\" file\n", LIST_FILENAME);
        return 5;
    }

    /************** CUDA **************/

    /* Pointer to the current slice of the list */
    int *host_int_list_ptr = &int_list[0];

    /* Pointer to the space for device copy of the list */
    int *cuda_int_list;

    /* Remaining ints to process */    
    int ints_remaining = num_of_ints;

    /* Start timer */
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    /* Allocate the space for device copy of the list-slice */
    hipMalloc((void **)&cuda_int_list, 
               num_of_cuda_blocks * num_of_cuda_threads * sizeof(int));

    /* As long as the number of remaining ints is equal or greater than the 
     * total number of parallel instances use all of them to compute */
    while (ints_remaining > 0)
    {
        /* Resolve the number of blocks and threads that will be used */
        resolve_cuda_threads(ints_remaining,
                             &num_of_cuda_blocks,
                             &num_of_cuda_threads);

        /* Copy the slice of the list to the device */
        hipMemcpy(cuda_int_list,
                   host_int_list_ptr,
                   num_of_cuda_blocks * num_of_cuda_threads * sizeof(int),
                   hipMemcpyHostToDevice);
                   
        /* Run the kernel on all threads */
        is_prime_kernel<<<num_of_cuda_blocks, 
                          num_of_cuda_threads>>>(cuda_int_list);

        /* Copy the results back to the host */
        hipMemcpy(host_int_list_ptr,
                   cuda_int_list,
                   num_of_cuda_blocks * num_of_cuda_threads * sizeof(int),
                   hipMemcpyDeviceToHost);

        /* Update the host list pointer */
        host_int_list_ptr += num_of_cuda_blocks * num_of_cuda_threads;

        /* Update the number of remaining ints */
        ints_remaining -= (num_of_cuda_blocks * num_of_cuda_threads);
    }

    /* Free the device's memory */
    hipFree(cuda_int_list);

    /* Write data to the file */
    for (int i = 0; i < num_of_ints; i++)
    {
        if (int_list[i] != -1)
        {
            prime_cnt++;
            fprintf(file_gen_ptr, "%d\n", int_list[i]);
        }
    }

    /* Put the number of primes int the first line of generated file */
    rewind(file_gen_ptr);
    fprintf(file_gen_ptr, "primes_found=%d(open-mp)\n", prime_cnt);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Elapsed: %f ms\n", time);
    printf("%d primes found...\n\r", prime_cnt);

    /**********************************/    

    /* Close the files and free the memory */    
    if (fclose(file_load_ptr) != 0)
    {
     	printf("Couldn't close \"%s\" file\n", filename);
        return 6;
    }

    if (fclose(file_gen_ptr) != 0)
    {
     	printf("Couldn't close \"%s\" file\n", LIST_FILENAME);
        return 7;
    }
    
    free(line_buff);
    free(int_list);

    printf("Done...\n");
    return 0;
}

/************************************ EOF *************************************/
